#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <algorithm>
#include <vector>
#include <numeric>
#define _USE_MATH_DEFINES
#include "math.h"

#ifdef _WIN32
#include<windows.h>
#endif
#include <GL/glew.h>
#include <GL/wglew.h>
#include <GL/glut.h>

const int N = 128;
const int uNumVoronoiPts = 16;
int ThreadsX = 4;
const int iterations = 100;
const double sim_rad = 1e18;

std::vector<float> results;
std::vector<float> vresults;

// Description of Voronoi Buf
struct VoronoiBuf
{
	double x;
	double y;
	float4 colour;
	void print()
	{
		std::cout << "\tP: " << x << " " << y << std::endl;
		std::cout << "\tC: " << colour.x << " " << colour.y << " " << colour.z << std::endl << std::endl;
	}
};

VoronoiBuf* Voronoi_d;

// Description of Body
struct body
{
	float4 colour;
	double4 position;
	double4 velocity;
	double2 force;
	bool operator==(body b)
	{
		return( (position.x == b.position.x) && (position.y == b.position.y) && (velocity.x == b.velocity.x) && (velocity.y == b.velocity.y) );
	}
	void error(body b)
	{
		double px = b.position.x - position.x;
		double py = b.position.y - position.y;
		double vx = b.velocity.x - velocity.x;
		double vy = b.velocity.y - velocity.y;
		double fx = b.velocity.z - velocity.z;
		double fy = b.velocity.w - velocity.w;

		std::cout << "Error in \n" << std::endl;
		std::cout << "\tP: " << px << " " << py;
		std::cout << std::endl;
		std::cout << "\tV: " << vx << " " << vy;
		std::cout << std::endl;
		std::cout << "\tF: " << fx << " " << fy;
		std::cout << std::endl;
	}
	void print()
	{
		std::cout << "\tM: " << position.w << " ";
		std::cout << "\tP: " << position.x << " " << position.y;
		std::cout << std::endl;
		std::cout << "\tV: " << velocity.x << " " << velocity.y;
		std::cout << "\tF: " << velocity.z << " " << velocity.w;
		std::cout << std::endl << std::endl;
	}
	void resetForce()
	{
		force = double2();
	}
	void addForce(body b)
	{
        double G = 6.67e-11;   // gravational constant
        double EPS = 3E4;      // softening parameter

        double dx = b.position.x - position.x;
        double dy = b.position.y - position.y;
        double dist = sqrt(dx*dx + dy*dy);
        double F = (G * position.w * b.position.w) / (dist*dist + EPS*EPS);
        force.x += F * dx / dist;
        force.y += F * dy / dist;
	}
	void update()
	{
        velocity.x += 1e10 * force.x / position.w;
        velocity.y += 1e10 * force.y / position.w;
        position.x += 1e10 * velocity.x;
        position.y += 1e10 * velocity.y;		
	}

};

double circlev(double x, double y)
{
	double solarmass = 1.98892e30;
	double r2 = sqrt(x*x + y*y);
	double numerator = (6.67e-11)*1e6*solarmass;
	return sqrt(numerator/r2);
}
double random()			{ return ((double)rand())/((double)RAND_MAX); }
template <typename T> int signum(T val) {
    return (T(0) < val) - (val < T(0));
}
body init_body(int i)
{
	double solarmass = 1.98892e30;
	if(i != 0)
	{
		double px = sim_rad*exp(-1.8)*(0.5 - random());
		double py = sim_rad*exp(-1.8)*(0.5 - random());
		double magv = circlev(px, py);

		double absangle = atan(abs(py/px));
		double thetav = M_PI/2 - absangle;
		double phiv = random() * M_PI;
		double vx = -1*signum(py)*cos(thetav)*magv;
		double vy = signum(px)*sin(thetav)*magv;

		if(random() < 0.5)
		{
			vx = -vx;
			vy = -vy;
		}

		double mass = random() * solarmass*10+1e20;

		body b;
		b.position.x = px;
		b.position.y = py;
		b.position.z = 0;
		b.position.w = mass;

		b.velocity.x = vx;
		b.velocity.y = vy;
		b.velocity.z = 0;
		b.velocity.w = 0;

		b.colour.x = 1.0f;
		b.colour.y = 0.0f;
		b.colour.z = 0.0f;
		b.colour.w = 1.0f;
		return b;
	}
	else
	{
		body b;
		b.position.x = 0;
		b.position.y = 0;
		b.position.z = 0;
		b.position.w = 1e6*solarmass;

		b.velocity.x = 0;
		b.velocity.y = 0;
		b.velocity.z = 0;
		b.velocity.w = 0;

		b.colour.x = 1.0f;
		b.colour.y = 1.0f;
		b.colour.z = 0.0f;
		b.colour.w = 1.0f;

		return b;
	}
}

	body* body_in;
	body* body_out;
	body* b_in;
	body* b_out;
	body* body_h;

void cudaQuery();

__device__ double2 bodyBodyInteraction(double4 bi, double4 bj, double2 a, bool output_thread)
{
    double G = 6.67e-11;   // gravational constant
    double EPS = 3E4;      // softening parameter

	// [2 FLOPS] 
	double dx = bj.x - bi.x;
	double dy = bj.y - bi.y;

	// [5 FLOPS]
	double dist = sqrt(dx*dx + dy*dy) + 0.0000125; // additional softening parameter
	
	//if(output_thread)
		//printf("dist - %g\n", dist);
	
	// [6 FLOPS]
	double F = (G * bi.w * bj.w) / (dist*dist + EPS*EPS);
	//if(output_thread)
		//printf("F - %g\n", F);

	// [6 FLOPS]
	a.x += F * dx / dist;
	a.y += F * dy / dist;

	return a;
}

__global__ void nbody_kernel(body* body_in, body* body_out)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx >= N)
		return;

	extern __shared__ double4 shared_position[];

	double4 position = body_in[idx].position;
		
	// calculate force contributions for body
	double2 force;
	force.x = 0.0;
	force.y = 0.0;

	for(int tile = 0; tile < gridDim.x; tile++)
	{
		unsigned int k = tile * blockDim.x + threadIdx.x;
		shared_position[threadIdx.x] = body_in[k].position;

		__syncthreads();


//#pragma unroll 128
		for(unsigned int counter = 0; counter < blockDim.x; counter++)
		{
			force = bodyBodyInteraction(position, shared_position[counter], force, false);
		}

		__syncthreads();
	}
	
	// Do update
	double4 velocity = body_in[idx].velocity;

	double4 v;
	v.x = velocity.x + 1e10 * force.x / position.w;
	v.y = velocity.y + 1e10 * force.y / position.w;
	v.z = force.x;
	v.w = force.y;

	double4 p;
	p.x = position.x + 1e10 * v.x;
	p.y = position.y + 1e10 * v.y;
	p.z = 0;
	p.w = position.w;

	body_out[idx].velocity = v;
	body_out[idx].position = p;
	body_out[idx].colour = body_in[idx].colour;
}

// Create Voronoi kernel
__global__ void create_voronoi( body* body_in, body* body_out, VoronoiBuf * v)
{
    // map from thread to pixel position
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
	double x = body_in[idx].position.x;
	double y = body_in[idx].position.y;

	// if in image
	// set min dist to distance from 1st position in buffer
	double d_x = (v[0].x - x);
	double d_y = (v[0].y - y);
	double d = (d_x*d_x + d_y*d_y);

	double minDist = d;
	int minDistPoint = 0;

	for(int i = 0; i < uNumVoronoiPts; i++)
	{
		double diff_x = (v[i].x - x);
		double diff_y = (v[i].y - y);
		double dist = (diff_x*diff_x + diff_y*diff_y);

		if(dist < minDist)
		{
			minDist = dist;
			minDistPoint = i;
		}
	}

	// now calculate the value at that position
	body_out[idx].colour = v[minDistPoint].colour;
	body_out[idx].position = body_in[idx].position;
	body_out[idx].velocity = body_in[idx].velocity;
}


bool cudaCheckAPIError(hipError_t err)
{
	if(err != hipSuccess)
	{
		std::cerr << "Error : " << hipGetErrorString(err) << std::endl;
		system("pause");
		return false;
	}

	return true;
}

template <class T>
void printLine(const char* message, T value)
{
	std::cout << message << "\t : " << value << std::endl;
}
template <class T>
void printLine(const char* message, T* value)
{
	if(value[2] == NULL)
		std::cout << message << "\t : " << value[0] << ", " << value[1] << std::endl;
	else
		std::cout << message << "\t : " << value[0] << " " << value[1] << " " << value[2] << std::endl;
}

void printBlank()
{
	std::cout << std::endl;
}

size_t RoundUp(int groupSize, int globalSize)
{
    int r = globalSize % groupSize;
    if(r == 0)
    {
        return globalSize;
    }
    else
    {
        return globalSize + groupSize - r;
    }
}

void cleanup()
{
	delete [] body_h;

	//cleanup
	cudaCheckAPIError( hipFree( body_in ) );
	cudaCheckAPIError( hipFree( body_out ) );

	cudaCheckAPIError( hipFree( b_in ) );
	cudaCheckAPIError( hipFree( b_out ) );

	cudaCheckAPIError( hipFree( Voronoi_d ) );

	exit(EXIT_SUCCESS);
}

void Key(unsigned char key, int x, int y)
{
    switch(key) 
    {
        case '\033': // escape quits
        case '\015': // Enter quits    
        case 'Q':    // Q quits
        case 'q':    // q (or escape) quits
            // Cleanup up and quit
                cleanup();
            break;
    }
}

float completeEvent(hipEvent_t start, hipEvent_t stop)
{
	// Add the stop event to the GPUs queue of work
	cudaCheckAPIError( hipEventRecord(stop, 0) );
	
	// Wait until the event has completed so it is safe to read
	cudaCheckAPIError( hipEventSynchronize(stop) );
	
	// Determine the time elapsed between the events
	float milliseconds = 0;
	cudaCheckAPIError( hipEventElapsedTime(&milliseconds, start, stop) );

	return milliseconds;
}

void checkBodies()
{
	body bodies[N];
	body check[N];

	// copy data from device to host

	//cudaCheckAPIError( hipEventRecord(startEvent, 0) );
		cudaCheckAPIError( hipMemcpy( bodies, body_in, sizeof(body)*N, hipMemcpyDeviceToHost ) );
	//completeEvent(startEvent, stopEvent, "retrieving output", false);
		cudaCheckAPIError( hipMemcpy( check, b_in, sizeof(body)*N, hipMemcpyDeviceToHost) );
 
	int errCount = 0;
	for(int i = 0; i < N; i++)
	{
		if(bodies[i] == check[i])
		{
		}
		else
		{
			//printf("\nERROR\n");
			//printf("original\n");
			//bodies[i].print();
			//printf("SM\n");
			//check[i].print();
			//printf("CPU\n");
			//body_h[i].print();

			//bodies[i].error(check[i]);

			//printf("\n\n");
			errCount++;
		}
	}
	if(errCount > 0)
		printf("Error count - %d\n\n", errCount);

	//system("pause");

}

void renderBodies(body* b)
{
	body bodies[N];

	// copy data from device to host
	//cudaCheckAPIError( hipEventRecord(startEvent, 0) );
		cudaCheckAPIError( hipMemcpy( bodies, b, sizeof(body)*N, hipMemcpyDeviceToHost ) );
	//completeEvent(startEvent, stopEvent, "retrieving output", false);

    glEnable( GL_POINT_SMOOTH );
    glEnable( GL_BLEND );
    glBlendFunc( GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA );
    glPointSize( 9.0 );

	glClearColor( 0.0, 0.0, 1.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	//glColor3f(1, 0, 0);

	glBegin(GL_POINTS);
		for(int i = 0; i < N; i++)
		{
			glColor3f( bodies[i].colour.x, bodies[i].colour.y, bodies[i].colour.z );	
			glVertex2f(bodies[i].position.x/(sim_rad/8), bodies[i].position.y/(sim_rad/8));
		}
	glEnd();

	glFinish();
	glutSwapBuffers();
}

// Execute voronoi kernel
void executeVoronoi()
{
	// Event parameters 
	hipEvent_t startEvent, stopEvent;

	// Create the event using hipEventCreate
	cudaCheckAPIError( hipEventCreate(&startEvent) );
	cudaCheckAPIError( hipEventCreate(&stopEvent) );

	dim3 grid( N/ThreadsX );
	dim3 block( ThreadsX );

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(create_voronoi), hipFuncCachePreferL1);

	cudaCheckAPIError( hipEventRecord(startEvent, 0) );
		create_voronoi <<< grid, block >>> (b_in, b_out, Voronoi_d);
	vresults.push_back( completeEvent(startEvent, stopEvent) );

	// Release events
	cudaCheckAPIError( hipEventDestroy(startEvent) );
	cudaCheckAPIError( hipEventDestroy(stopEvent) );
}

void nBodySM()
{
	// Event parameters 
	hipEvent_t startEvent, stopEvent;

	// Create the event using hipEventCreate
	cudaCheckAPIError( hipEventCreate(&startEvent) );
	cudaCheckAPIError( hipEventCreate(&stopEvent) );

	dim3 grid(RoundUp(ThreadsX, N)/ThreadsX);
	dim3 block(ThreadsX);

	// compute body updates
	cudaCheckAPIError( hipEventRecord(startEvent, 0) );
		nbody_kernel <<< grid, block, ThreadsX*sizeof(double4) >>> (b_out, b_in);
	results.push_back( completeEvent(startEvent, stopEvent) );

	// Release events
	cudaCheckAPIError( hipEventDestroy(startEvent) );
	cudaCheckAPIError( hipEventDestroy(stopEvent) );
}

void outputStats(std::vector<float>& results)
{
	// Median	
	std::sort( results.begin(), results.end());
	double med = 0.0;
	if(results.size()/2 == 0)
		med = results[ results.size()/2 ];
	else
	{
		med = (results[ results.size()/2 ] + results[ results.size()/2 - 1])/2.0; 
	}
	printf("Median: %.2f ms\n", med);
	//printf("\t %.2f Mop/s\n", computeStats(med));
/*
	// Mean
	double sum = std::accumulate(std::begin(results), std::end(results), 0.0);
	double m =  sum / results.size();
	printf("Mean: %.2f ms\n", m);
	//printf("\t %.2f Mop/s\n", computeStats(m));

	// Standard deviation
	double accum = 0.0;
	std::for_each (std::begin(results), std::end(results), [&](const double d) {
		accum += (d - m) * (d - m);
	});
	double stdev = sqrt(accum / (results.size()-1));
	printf("Standard Deviation: %.2f\n", stdev);*/

	//printf("1. %.2f %d. %.2f\n", results[0], results.size()-1, results[results.size()-1]);
	results.clear();
}

void Draw()
{	
	executeVoronoi();

	nBodySM();
	
	// swap buffers
//	std::swap(body_in, body_out);
//	std::swap(b_in, b_out);

	//checkBodies();
	renderBodies(b_in);


	static int i = 0;
	i++;
	if(i > iterations)
	{
		i = 0;
		
		// Output Interaction Results
		printf("\nVoronoi NBody Results\n");
		printf("Threads -\t%d\n", ThreadsX);
		printf("Voronoi -\t");
		outputStats(vresults);
		printf("NBody -\t");
		outputStats(results);

		ThreadsX *= 2;

	//	system("pause");
	}

	if((ThreadsX > N) || (ThreadsX > 1024))
	{
		system("pause");
		cleanup();
	}
	
}

void initGL(int argc, char *argv[], int wWidth, int wHeight)
{
	// init gl
	glutInit( &argc, argv );
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowPosition(500, 100);
	glutInitWindowSize( wWidth, wHeight );
	glutCreateWindow( "CUDA Nbody" );

	// set callback functions
	glutKeyboardFunc(Key);
	glutDisplayFunc(Draw);
    glutIdleFunc(Draw);
   
   glewInit();
    if (glewIsSupported("GL_VERSION_2_1"))
        printf("Ready for OpenGL 2.1\n");
    else 
         printf("Warning: Detected that OpenGL 2.1 not supported\n");

	//wglSwapIntervalEXT(false);
}

int main(int argc, char** argv)
{
	printf("N Body Benchmark CUDA\n\n");

	// Initial body data
	const int body_size = sizeof(body)*N;
	const int voronoi_size = sizeof(VoronoiBuf)*uNumVoronoiPts;

	printf("Body List\n");
	body_h = (body*)malloc( body_size );
	for(int i = 0; i < N; i++)
	{
		body_h[i] = init_body(i);
		//body_h[i].print();
	}
	printf("\n");

	// Generate Voronoi Points	
	VoronoiBuf* Voronoi_h = (VoronoiBuf*)malloc(voronoi_size);
	printf("Program Data\n");
	printf("Number of Voronoi Points :\t%d\n", uNumVoronoiPts);
	int k = 0;
	int dim = sqrt((float)uNumVoronoiPts);
	double spacing_x = (sim_rad/4)/ dim;
	double spacing_y = (sim_rad/4)/ dim;
	printf("%d %g %g\n\n", dim, spacing_x, spacing_y); 
	printf("Voronoi Points\n");
	for(int i = -1*dim/2; i < dim/2; i++)
	{
		for(int j = -1*dim/2; j < dim/2; j++)
		{
			Voronoi_h[k].x = spacing_x/2 + spacing_x*i;
			Voronoi_h[k].y = spacing_y/2 + spacing_y*j;
			Voronoi_h[k].colour.x = (sin(2.4*k + 0) *127 + 128)/255;
			Voronoi_h[k].colour.y = (sin(2.4*k + 2) *127 + 128)/255;
			Voronoi_h[k].colour.z = (sin(2.4*k + 4) *127 + 128)/255;
			Voronoi_h[k].colour.w = 1.0f;
			//Voronoi_h[k].print();
			k++;
		}
	}

	// allocate memory on device for buffers
	cudaCheckAPIError( hipMalloc( (void**)&body_in, body_size) );
	cudaCheckAPIError( hipMalloc( (void**)&body_out, body_size) );
	cudaCheckAPIError( hipMalloc( (void**)&b_in, body_size) );
	cudaCheckAPIError( hipMalloc( (void**)&b_out, body_size) );
	cudaCheckAPIError( hipMalloc( (void**)&Voronoi_d, voronoi_size) );


	// copy data from host to device
	cudaCheckAPIError( hipMemcpy( body_in, body_h, body_size, hipMemcpyHostToDevice) );
	cudaCheckAPIError( hipMemcpy( b_in, body_h, body_size, hipMemcpyHostToDevice) ); //same intial conditions
	cudaCheckAPIError( hipMemcpy( Voronoi_d, Voronoi_h, voronoi_size, hipMemcpyHostToDevice) );
	free( Voronoi_h );

	
	// Output some useful data
	printf("Number of Bodies : \t%d\n", N);

	printf("NBody\n");
	printf("Global Work Size :\t%d\n", RoundUp(ThreadsX, N)/ThreadsX );
	printf("Local Work Size :\t%d\n\n\n", ThreadsX);

		initGL(argc, argv, 512, 512);
        glutMainLoop();

	return 0;
}



// query device properties
void cudaQuery()
{
	// determine number of CUDA devices
	int count;
	cudaCheckAPIError( hipGetDeviceCount(&count) );
	printLine("Number of CUDA Devices ", count);
	printBlank();

	// output information on all devices
	for(int i = 0; i < count; i++)
	{
		printLine("Device ", i+1);

		// determine properties
		hipDeviceProp_t properties;
		cudaCheckAPIError( hipGetDeviceProperties(&properties, i) );

		printLine("Name			", &properties.name);
		printLine("Total Global Mem	", properties.totalGlobalMem);
		printLine("Shared Mem Per Block	", properties.sharedMemPerBlock);
		printLine("Regs Per Block		", properties.regsPerBlock);
		printLine("Warp Size		", properties.warpSize);
		printLine("MemPitch		", properties.memPitch);
		printLine("Max Threads Per Block	", properties.maxThreadsPerBlock);
		printLine("Max Threads Dim		", properties.maxThreadsDim);
		printLine("Max Grid Size		", properties.maxGridSize);
		printLine("Total Const Mem		", properties.totalConstMem);
		printLine("Major			", properties.major);
		printLine("Minor			", properties.minor);
		printLine("Clock Rate		", properties.clockRate);
		printLine("Texture Alignment	", properties.textureAlignment);
		printLine("Device Overlap		", properties.deviceOverlap);
		printLine("Multi Processor Count	", properties.multiProcessorCount);
		printLine("Kernel Exec Timeout Enabled", properties.kernelExecTimeoutEnabled);
		printLine("Integrated		", properties.integrated);
		printLine("Can Map Host Memory	", properties.canMapHostMemory);
		printLine("Compute Mode		", properties.computeMode);
		printLine("Max Texture 1D		", properties.maxTexture1D);
		printLine("Max Surface 2D		", properties.maxSurface2D);
		printLine("Max Texture 2D		", properties.maxTexture2D);
		printLine("Max Texture 3D		", properties.maxTexture3D);
		printLine("Concurrent Kernels	", properties.concurrentKernels);
	}
	printBlank();
}
